#include "hip/hip_runtime.h"
#include "push.h"
//#define LOG
inline __device__
void interpolate_E(Grid grid, int ix, int iy, int iz, part_t* position, double* target)
{
    int nx = grid.nx, ny = grid.ny, nz = grid.nz;
    double dx = grid.dx, dy = grid.dy, dz = grid.dz;
    //assume energy conserving
    part_t xdiff = (position[0]- ix*dx)/dx;
    part_t ydiff = (position[1]-iy*dy)/dy;
    part_t zdiff = (position[2] - iz*dz)/dz;
    int not_lastx = ix != nx-1;
    int not_lasty = iy != ny-1;
    int not_lastz = iz != nz-1;
    double a[] = {1-xdiff, xdiff, 1-ydiff, ydiff, 1-zdiff, zdiff};
    double SE[] = {a[2*1] * a[2*2],     a[2*0] * a[2*2]    ,   a[2*0] * a[2*1],
                   0,                   a[2*0+1] * a[2*2]  , a[2*0+1] * a[2*1],
                   a[2*1+1] * a[2*2],   0                  , a[2*0] * a[2*1+1],
                   0,                   0                  ,   a[2*0+1] * a[2*1+1],
                   a[2*1] * a[2*2+1],   a[2*0] * a[2*2+1]  , 0,
                   0,                   a[2*0+1] * a[2*2+1], 0,
                   a[2*1+1] * a[2*2+1], 0,                   0,
                   0,                   0,                   0};
    //iterate for E in X direction
    target[0] += SE[0*3+0]*Ex_cu(ix- !not_lastx, iy, iz);
    target[0] += SE[2*3+0]*Ex_cu(ix- !not_lastx, iy+not_lasty, iz)*not_lasty;
    target[0] += SE[4*3+0]*Ex_cu(ix- !not_lastx, iy, iz+not_lastz)*not_lastz;
    target[0] += SE[6*3+0]*Ex_cu(ix- !not_lastx, iy+not_lasty, iz+not_lastz)*not_lasty*not_lastz;
    target[1] += SE[0*3+1]*Ey_cu(ix, iy- !not_lasty, iz);
    target[1] += SE[1*3+1]*Ey_cu(ix+not_lastx, iy- !not_lasty, iz)*not_lastx;
    target[1] += SE[4*3+1]*Ey_cu(ix, iy- !not_lasty, iz+not_lastz)*not_lastz;
    target[1] += SE[5*3+1]*Ey_cu(ix+not_lastx, iy- !not_lasty, iz+not_lastz)*not_lastx*not_lastz;
    target[2] += SE[0*3+2]*Ez_cu(ix, iy, iz-!not_lastz);
    target[2] += SE[1*3+2]*Ez_cu(ix+not_lastx, iy, iz-!not_lastz)*not_lastx;
    target[2] += SE[2*3+2]*Ez_cu(ix, iy+not_lasty, iz-!not_lastz)*not_lasty;
    target[2] += SE[3*3+2]*Ez_cu(ix+not_lastx, iy+not_lasty, iz-!not_lastz)*not_lastx*not_lasty;
};
inline __device__
void interpolate_B(Grid grid, int ix, int iy, int iz, double* position, double* target)
{
    int nx = grid.nx, ny = grid.ny, nz = grid.nz;
    double dx = grid.dx, dy = grid.dy, dz = grid.dz;
    //DEFINE_GRID_CONSTANTS;
    int not_lastx = ix != nx-1;
    int not_lasty = iy != ny-1;
    int not_lastz = iz != nz-1;
    //assume energy conserving
    part_t xdiff = (position[0]- ix*dx)/dx;
    part_t ydiff = (position[1]-iy*dy)/dy;
    part_t zdiff = (position[2] - iz*dz)/dz;
    double a[] = {1-xdiff, xdiff, 1-ydiff, ydiff, 1-zdiff, zdiff};
    double SB[] = {a[2*0]  , a[2*1]  , a[2*2],
                   a[2*0+1], 0       , 0,
                   0       , a[2*1+1], 0,
                   0       , 0       , 0,
                   0       , 0       , 0,
                   0       , 0       , a[2*2+1],
                   0       , 0       , 0,
                   0       , 0       , 0};
    double mu_loc = mu0;//MU(ix, iy, iz);
    //printf("Target %e %e %e\n", target[0], target[1], target[2]);
    //add all B components
    target[0] += SB[0*3+0]*Hx_cu(ix, iy-!not_lasty, iz-!not_lastz)*mu_loc;
    target[0] += SB[1*3+0]*Hx_cu(ix+not_lastx, iy-!not_lasty, iz-!not_lastz)*mu_loc*not_lastx;
    target[1] += SB[0*3+1]*Hy_cu(ix-!not_lastx, iy, iz-!not_lastz)*mu_loc;
    target[1] += SB[2*3+1]*Hy_cu(ix-!not_lastx, iy+not_lasty, iz-!not_lastz)*mu_loc*not_lasty;
    target[2] += SB[0*3+2]*Hz_cu(ix-!not_lastx, iy-!not_lasty, iz)*mu_loc;
    target[2] += SB[5*3+2]*Hz_cu(ix-!not_lastx, iy-!not_lasty, iz+not_lastz)*mu_loc*not_lastz;
};
#define TRILIN_INTERP(x, y, z, xa, ya, za) {\
    xd = 1-fabs((x)-(xa))/dx;\
    yd = 1-fabs((y)-(ya))/dy;\
    zd = 1-fabs((z)-(za))/dz;\
    weight = xd*yd*zd;\
};
__global__
void particle_push_cu(Grid grid, double step, part_t* logger, int stepnum, int max_particles) {
    //update each particle according to the fields at its current position
    //will want to seperate the field gather and the particle push steps later, for now focus on basic procedure
    int base_index = (blockIdx.x*blockDim.x + threadIdx.x)/max_particles;//base particle
    int numparticles = grid.num_particles;
    if (base_index >= numparticles) {
        return;
    }
    part_t m = grid.mass_p;
    space_t dx = grid.dx, dy = grid.dy, dz = grid.dz;
    //int nx = grid.nx, ny = grid.ny, nz = grid.nz;
    space_t xmin = grid.xlims[0], ymin = grid.ylims[0], zmin = grid.zlims[0];
    part_t q = grid.q_p;
    Particle* p_arr = grid.particles;
    int p_index;
    int nx = grid.nx;
    int ny = grid.ny;
    int nz = grid.nz;
    for(p_index = base_index; p_index < base_index + max_particles; p_index++) {

        Particle* p = &p_arr[p_index];
        part_t px, py, pz, x, y, z;
        px = p->px;
        py = p->py;
        pz = p->pz;
        x = p->x;
        y = p->y;
        z = p->z;
        double next_x = x + (px/m)*step;
        double next_y = y + (py/m)*step;
        double next_z = z + (pz/m)*step;
        int condx_max, condx_min, condy_max, condy_min, condz_max, condz_min;
        condx_max = (next_x >= grid.xlims[1]);
        condx_min = (next_x <= grid.xlims[0]);
        condy_max = (next_y >= grid.ylims[1]);
        condy_min = (next_y <= grid.ylims[0]);
        condz_max = (next_z >= grid.zlims[1]);
        condz_min = (next_z <= grid.zlims[0]);
        x = next_x*(!condx_max && !condx_min) + condx_max*(grid.xlims[0] + 1e-10) + condx_min*(grid.xlims[1] - 1e-10);
        y = next_y*(!condy_max && !condy_min) + condy_max*(grid.ylims[0] + 1e-10) + condy_min*(grid.ylims[1] - 1e-10);
        z = next_z*(!condz_max && !condz_min) + condz_max*(grid.zlims[0] + 1e-10) + condz_min*(grid.zlims[1] - 1e-10);
        p->x = x;
        p->y = y;
        p->z = z;
        field_t elocal[] = {0.0, 0.0, 0.0};
        field_t blocal[] = {0.0, 0.0, 0.0};
        part_t pos[3] = {x, y, z};
        int ix, iy, iz;
        ix = (int)((x - xmin)/dx);
        iy = (int)((y - ymin)/dy);
        iz = (int)((z - zmin)/dz);
        interpolate_E(grid, ix, iy, iz, pos, elocal);
        interpolate_B(grid, ix, iy, iz, pos, blocal);
        part_t qconst = q*step/2;
        part_t p_temp[] = {
            px+elocal[0]*qconst,
            py+elocal[1]*qconst,
            pz+elocal[2]*qconst
        };
        part_t t_vec[] = {
            blocal[0]*qconst,
            blocal[1]*qconst,
            blocal[2]*qconst
        };

        part_t p_prime[] = {
            p_temp[0] + (p_temp[1]*t_vec[2] - p_temp[2]*t_vec[1]),
            p_temp[1] + (p_temp[2]*t_vec[0] - p_temp[0]*t_vec[2]),
            p_temp[2] + (p_temp[0]*t_vec[1] - p_temp[1]*t_vec[0])
        };
        part_t tmag = sqrt(t_vec[0]*t_vec[0] + t_vec[1]*t_vec[1] + t_vec[2]*t_vec[2]);
        part_t tconst = 2/(tmag*tmag + 1);
        p_temp[0] += tconst*(p_prime[1]*t_vec[2] - p_prime[2]*t_vec[1]);
        p_temp[1] += tconst*(p_prime[2]*t_vec[0] - p_prime[0]*t_vec[2]);
        p_temp[2] += tconst*(p_prime[0]*t_vec[1] - p_prime[1]*t_vec[0]);
        //particle push with Boris step
        p->px = p_temp[0] + elocal[0]*qconst;
        p->py = p_temp[1] + elocal[1]*qconst;
        p->pz = p_temp[2] + elocal[2]*qconst;
        double vx = (p_temp[0] + elocal[0]*qconst)/m;
        double vy = (p_temp[1] + elocal[1]*qconst)/m;
        double vz = (p_temp[1] + elocal[1]*qconst)/m;
        #ifdef LOG
            logger[(stepnum*numparticles + p_index)*3] = x;
            logger[(stepnum*numparticles + p_index)*3 + 1] = y;
            logger[(stepnum*numparticles + p_index)*3 + 2] = z;
        #endif
        int not_lastx = (ix != nx-1);
        int not_lasty = (iy != ny-1);
        int not_lastz = (iz != nz-1);
        double xg[] = {dx*ix + xmin, dx*(ix+1) + xmin};
        double yg[] = {dy*iy + ymin, dy*(iy+1) + ymin};
        double zg[] = {dz*iz + zmin, dz*(iz+1) + zmin};
        int ox = ix, oy = iy, oz = iz;
        double xd, yd, zd, weight;
        double jx_p = vx*q;
        double jy_p = vy*q;
        double jz_p = vz*q;
        //ix iy iz
        TRILIN_INTERP(x, y, z, xg[ox-ix], yg[oy-iy], zg[oz-iz]);

        atomicAdd(&Jx_cu(ox, oy,  oz),  jx_p*weight);
        atomicAdd(&Jy_cu(ox, oy,  oz),  jy_p*weight);
        atomicAdd(&Jz_cu(ox, oy,  oz),  jz_p*weight);
        atomicAdd(&RHO_cu(ox, oy,  oz),  weight*q);
        //ix+1 iy iz
        ox = ix + not_lastx;
        TRILIN_INTERP(x, y, z, xg[ox-ix], yg[oy-iy], zg[oz-iz]);
        atomicAdd(&Jx_cu(ox, oy,  oz),  jx_p*weight*(not_lastx));
        atomicAdd(&Jy_cu(ox, oy,  oz),  jy_p*weight*(not_lastx));
        atomicAdd(&Jz_cu(ox, oy,  oz),  jz_p*weight*(not_lastx));
        atomicAdd(&RHO_cu(ox, oy,  oz),  weight*q);
        //ix+1 iy+1 iz
        oy = iy + not_lasty;
        TRILIN_INTERP(x, y, z, xg[ox-ix], yg[oy-iy], zg[oz-iz]);
        atomicAdd(&Jx_cu(ox, oy,  oz),  jx_p*weight*(not_lastx)*(not_lasty));
        atomicAdd(&Jy_cu(ox, oy,  oz),  jy_p*weight*(not_lastx)*(not_lasty));
        atomicAdd(&Jz_cu(ox, oy,  oz),  jz_p*weight*(not_lastx)*(not_lasty));
        atomicAdd(&RHO_cu(ox, oy,  oz),  weight*q);
        //ix+1 iy iz+1
        oy = iy;
        oz = iz + not_lastz;
        TRILIN_INTERP(x, y, z, xg[ox-ix], yg[oy-iy], zg[oz-iz]);
        atomicAdd(&Jx_cu(ox, oy,  oz),  jx_p*weight*(not_lastx)*(not_lastz));
        atomicAdd(&Jy_cu(ox, oy,  oz),  jy_p*weight*(not_lastx)*(not_lastz));
        atomicAdd(&Jz_cu(ox, oy,  oz),  jz_p*weight*(not_lastx)*(not_lastz));
        atomicAdd(&RHO_cu(ox, oy,  oz),  weight*q);
        //ix iy+1 iz+1
        ox = ix;
        oy = iy + not_lasty;
        TRILIN_INTERP(x, y, z, xg[ox-ix], yg[oy-iy], zg[oz-iz]);
        atomicAdd(&Jx_cu(ox, oy,  oz),  jx_p*weight*(not_lastz)*(not_lasty));
        atomicAdd(&Jy_cu(ox, oy,  oz),  jy_p*weight*(not_lastz)*(not_lasty));
        atomicAdd(&Jz_cu(ox, oy,  oz),  jz_p*weight*(not_lastz)*(not_lasty));
        atomicAdd(&RHO_cu(ox, oy,  oz),  weight*q);
        //ix+1 iy+1 iz+1
        ox = ix + not_lastx;
        TRILIN_INTERP(x, y, z, xg[ox-ix], yg[oy-iy], zg[oz-iz]);
        atomicAdd(&Jx_cu(ox, oy,  oz),  jx_p*weight*(not_lastx)*(not_lasty)*(not_lastz));
        atomicAdd(&Jy_cu(ox, oy,  oz),  jy_p*weight*(not_lastx)*(not_lasty)*(not_lastz));
        atomicAdd(&Jz_cu(ox, oy,  oz),  jz_p*weight*(not_lastx)*(not_lasty)*(not_lastz));
        atomicAdd(&RHO_cu(ox, oy,  oz),  weight*q);
    }
}
#define TpB 256
#define TMAX 1000000

void gpu_particle_push(Grid* grid, double step, part_t* logger, int stepnum) {
    int blocks, max_particles;
    if (NP > TMAX) {
        max_particles = ceil((double)NP/TMAX);
        blocks = TMAX/(TpB) + 1;
    }else{
        max_particles = 1;
        blocks = NP/(TpB) + 1;
    }
    particle_push_cu<<<blocks, TpB>>>(*grid, step, logger, stepnum, max_particles);
    hipError_t stat = hipGetLastError();
    if(stat != hipSuccess) {
        printf("%s\n", hipGetErrorString(stat));
        getchar();
    }
    hipDeviceSynchronize();
}